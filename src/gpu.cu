#include "hip/hip_runtime.h"
#include "util.hpp"
#include <hip/hip_runtime.h>
#include <vector>

using std::vector;

__global__
void add_one(uint32_t* data, unsigned int len) {
	for (unsigned int i=0; i<len; i++) {
		data[i] += 1;
	}
}

// adapted from https://stackoverflow.com/a/14038590
#define gpu_assert(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace gpu {
void sort(vector<uint32_t>& data) {
	const auto bytes = data.size()*sizeof(uint32_t);
	uint32_t* d_data = nullptr;
	const uint32_t* h_data = data.data();
	auto ok = hipMalloc((void**) &d_data, bytes);
	gpu_assert(ok);

	ok = hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
	gpu_assert(ok);

	add_one<<<1, 1>>>(d_data, data.size());

	ok = hipDeviceSynchronize();
	gpu_assert(ok);

	ok = hipMemcpy(d_data, h_data, bytes, hipMemcpyDeviceToHost);
	gpu_assert(ok);

	hipFree((void*)h_data);
}
}
