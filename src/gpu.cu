#include "util.hpp"
#include <hip/hip_runtime.h>
#include <vector>

using std::vector;

void test(int hi) {
};

// namespace gpu {
void sort(vector<uint32_t>& data) {
	auto bytes = data.size()*sizeof(uint32_t);
	uint32_t* d_data = nullptr;
	uint32_t* h_data = data.data();
	auto alloc_ok = hipMalloc((void**) &d_data, bytes);
	assert(alloc_ok==true || "could not allocate memory on gpu");
	auto copy_ok = hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
	assert(copy_ok==true || "could not copy data to gpu");

	hipFree(h_data);
}
// }
